#include "hip/hip_runtime.h"
// Simple CUDA example by Ingemar Ragnemalm 2009. Simplest possible?
// Assigns every element in an array with its index.

// nvcc simple.cu -L /usr/local/cuda/lib -lcudart -o simple

#include <stdio.h>
#include <math.h>

const int N = 16; 
const int blocksize = 16; 

__global__ 
void simple(float *c) 
{
	c[threadIdx.x] = sqrt(c[threadIdx.x]);
}

int main()
{
	float *c = new float[N];
	float *o = new float[N];
	for(int i =0; i<N;++i)
	{
		c[i] = i*2+3.14568*i;
	}

	
	float *cd;
	const int size = N*sizeof(float);
	
	hipMalloc( (void**)&cd, size );
	hipMemcpy( cd, c, size, hipMemcpyHostToDevice ); 

	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	simple<<<dimGrid, dimBlock>>>(cd);
	hipDeviceSynchronize();
	hipMemcpy( o, cd, size, hipMemcpyDeviceToHost ); 
	hipFree( cd );
	
	/*for (int i = 0; i < N; i++)
		printf("%f ", c[i]);
	printf("\n");*/

	for (int i = 0; i < N; i++){
		printf("%f %f",o[i], sqrt(c[i]));
		if(o[i] == sqrt(c[i])) printf(" OK\n");
		else printf(" DIFFERENT \n");
	}
	printf("\n");

	delete[] c;
	printf("done\n");
	return EXIT_SUCCESS;
}
